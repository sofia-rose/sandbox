
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
  int size = n * sizeof(float);
  float *A_d, *B_d, *C_d;
  hipError_t err;

  err = hipMalloc((void **) &A_d, size);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipMalloc((void **) &B_d, size);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipMalloc((void **) &C_d, size);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }


  // Launch ceil(n/256) blocks of 256 threads each
  vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);


  err = hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipFree(A_d);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  err = hipFree(B_d);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  err = hipFree(C_d);
  if (hipSuccess != err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
}

int main() {
  float *A, *B, *C;
  int n = 1024*1024;
  size_t size = n * sizeof(float);

  A = (float*)malloc(size);
  if (NULL == A) {
    printf("Cannot allocate memory for vector A\n");
    exit(EXIT_FAILURE);
  }

  B = (float*)malloc(size);
  if (NULL == B) {
    printf("Cannot allocate memory for vector B\n");
    exit(EXIT_FAILURE);
  }

  C = (float*)malloc(size);
  if (NULL == B) {
    printf("Cannot allocate memory for vector C\n");
    exit(EXIT_FAILURE);
  }

  for (int i = 0; i < n; ++i) {
    A[i] = i;
    B[i] = i;
  }

  vecAdd(A, B, C, n);

  for (int i = 0; i < n; ++i) {
    printf("A[%d] + B[%d] = C[%d]: %f + %f = %f\n", i, i, i, A[i], B[i], C[i]);
  }

  return 0;
}
